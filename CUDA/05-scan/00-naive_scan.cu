
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    const int block_size = 1024;

    const int array_size = 1 << 20;
    int* h_array = new int[array_size];
    for (int i = 0; i < array_size; ++i) {
        h_array[i] = 1;
    }

    int* output = new int[array_size];

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);

    output[0] = h_array[0];
    for (int i = 1; i < array_size; ++i) {
        output[i] = output[i - 1] + h_array[i];
    }


    hipEventRecord(stop);


    hipEventSynchronize(stop);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << " elapsed" << std::endl;

    std::cout << output[array_size - 1] << std::endl;

    delete[] h_array;
    delete[] output;


}
